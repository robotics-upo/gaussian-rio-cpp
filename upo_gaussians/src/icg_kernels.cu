#include "hip/hip_runtime.h"
#include "gaussian_scan_matching.hpp"

#define LAUNCH_ARGS <<<dim3(m_numBlocks, m_numParticles), UPO_CUDA_NUM_THREADS>>>

namespace upo_gaussians::detail {

namespace {

__global__ void icgMatchupP2G_Impl(
	uint32_t N_points,
	uint32_t N_gaussians,
	float max_sqmahal,
	Vecf<4> const* in_T_tran,
	Quatf const* in_T_rot,
	Vecf<4> const* in_points,
	Vecf<4> const* in_g_centers,
	Vecf<4> const* in_g_invscale,
	Quatf const* in_g_invrot,
	int32_t* out_matchups,
	MatchOut* out_grid_matchout
)
{
	auto block_idx = blockIdx.x + gridDim.x * blockIdx.y;
	auto transform_idx = blockIdx.y;
	auto point_idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto lane_idx = threadIdx.x & (CUDA_WARP_SIZE - 1);
	auto warp_idx = threadIdx.x / CUDA_WARP_SIZE;

	bool is_active = point_idx < N_points;
	int32_t best_idx = -1;
	float best_sqmahal = is_active ? max_sqmahal : 0.0f;

	Vecf<3> point;
	if (is_active) {
		point = in_T_tran[transform_idx].segment<3>(0) + in_T_rot[transform_idx]*in_points[point_idx].segment<3>(0);
	}

	for (uint32_t base = 0; base < N_gaussians; base += CUDA_WARP_SIZE) {
		unsigned maxfetch = capToWarpSize(N_gaussians - base);

		Vecf<3> warp_g_center;
		Vecf<3> warp_g_invscale;
		Quatf   warp_g_invrot;
		if (lane_idx < maxfetch) {
			warp_g_center   = in_g_centers[base+lane_idx].segment<3>(0);
			warp_g_invscale = in_g_invscale[base+lane_idx].segment<3>(0);
			warp_g_invrot   = in_g_invrot[base+lane_idx];
		}

		for (unsigned i = 0; i < maxfetch; i ++) {
			Vecf<3> g_center;
			Vecf<3> g_invscale;
			Quatf   g_invrot;
			for (unsigned row = 0; row < 3; row ++) {
				g_center(row) = __shfl_sync(UINT32_MAX, warp_g_center(row), i);
				g_invscale(row) = __shfl_sync(UINT32_MAX, warp_g_invscale(row), i);
				g_invrot.coeffs()(row) = __shfl_sync(UINT32_MAX, warp_g_invrot.coeffs()(row), i);
			}
			g_invrot.coeffs()(3) = __shfl_sync(UINT32_MAX, warp_g_invrot.coeffs()(3), i);

			if (!is_active) {
				continue;
			}

			int32_t curidx = base+i;
			auto vector = g_invscale.cwiseProduct(g_invrot*(point - g_center));
			float sqmahal = vector.squaredNorm();

			if (sqmahal < best_sqmahal) {
				best_idx = curidx;
				best_sqmahal = sqmahal;
			}
		}
	}

	if (is_active) {
		out_matchups[point_idx + N_points*transform_idx] = best_idx;
	}

	// Sum-reduce for this warp
	uint32_t matched_mask = __ballot_sync(UINT32_MAX, best_idx >= 0);
	for (unsigned w = CUDA_WARP_SIZE/2; w; w >>= 1) {
		best_sqmahal += __shfl_xor_sync(UINT32_MAX, best_sqmahal, w);
	}

	// Prepare the thread block sum-reduce
	__shared__ float block_sqmahal[UPO_CUDA_WARPS_PER_BLOCK];
	__shared__ int32_t block_matches[UPO_CUDA_WARPS_PER_BLOCK];
	if (lane_idx == 0) {
		block_sqmahal[warp_idx] = best_sqmahal;
		block_matches[warp_idx] = __builtin_popcount(matched_mask);
	}

	// Sum-reduce for this thread block
	for (unsigned w = UPO_CUDA_WARPS_PER_BLOCK/2; w; w >>= 1) {
		__syncthreads();
		if (lane_idx == 0 && warp_idx < w) {
			block_sqmahal[warp_idx] += block_sqmahal[warp_idx+w];
			block_matches[warp_idx] += block_matches[warp_idx+w];
		}
	}

	// Store the reduced value
	if (lane_idx == 0 && warp_idx == 0) {
		auto& out = out_grid_matchout[block_idx];
		out.sqmahal = block_sqmahal[0];
		out.matches = block_matches[0];
	}
}

__global__ void icgSumReducePxy_Impl(
	uint32_t N_points,
	int32_t const* in_matchups,
	Vecf<4> const* in_points,
	Vecf<4> const* in_g_centers,
	Vecf<4> const* in_g_invscale,
	Quatf const* in_g_invrot,
	symposmat3* out_grid_mats,
	MultiVecf<4,2>* out_grid_xy
)
{
	auto block_idx = blockIdx.x + gridDim.x * blockIdx.y;
	auto transform_idx = blockIdx.y;
	auto point_idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto lane_idx = threadIdx.x & (CUDA_WARP_SIZE - 1);
	auto warp_idx = threadIdx.x / CUDA_WARP_SIZE;

	// Fetch the index of the Gaussian associated to this thread
	int32_t g_idx = -1;
	if (point_idx < N_points) {
		g_idx = in_matchups[point_idx + N_points*transform_idx];
	}

	// Compute this thread's matrix
	symposmat3 P;
	MultiVecf<3,2> xy = MultiVecf<3,2>::Zero();
	if (g_idx >= 0) {
		Matf<3> M = in_g_invscale[g_idx].segment<3>(0).asDiagonal() * in_g_invrot[g_idx].toRotationMatrix();
		P = Matf<3>(M.transpose() * M);
		xy.col(0) = M.transpose() * (M * in_points[point_idx].segment<3>(0));
		xy.col(1) = M.transpose() * (M * in_g_centers[g_idx].segment<3>(0));
	}

	// Sum-reduce this warp's matrix
	for (unsigned w = CUDA_WARP_SIZE/2; w; w >>= 1) {
		for (unsigned i = 0; i < 2; i ++) {
			for (unsigned j = 0; j < 3; j ++) {
				P(i,j) += __shfl_xor_sync(UINT32_MAX, P(i,j), w);
				xy(j,i) += __shfl_xor_sync(UINT32_MAX, xy(j,i), w);
			}
		}
	}

	// Store each warp's matrix into sharedmem
	__shared__ Wrapper<decltype(P)>  block_P [UPO_CUDA_WARPS_PER_BLOCK];
	__shared__ Wrapper<decltype(xy)> block_xy[UPO_CUDA_WARPS_PER_BLOCK];
	if (lane_idx == 0) {
		*block_P[warp_idx] = P;
		*block_xy[warp_idx] = xy;
	}

	// Sum-reduce this thread block's matrix
	for (unsigned w = UPO_CUDA_WARPS_PER_BLOCK/2; w; w >>= 1) {
		__syncthreads();
		if (lane_idx == 0 && warp_idx < w) {
			*block_P[warp_idx] += *block_P[warp_idx+w];
			*block_xy[warp_idx] += *block_xy[warp_idx+w];
		}
	}

	// Store this thread block's reduced matrix
	if (lane_idx == 0 && warp_idx == 0) {
		out_grid_mats[block_idx] = *block_P[0];
		out_grid_xy[block_idx].block(0,0,3,2) = *block_xy[0];
	}
}

__global__ void icgSumReduceRotOpt_Impl(
	uint32_t N_points,
	int32_t const* in_matchups,
	MultiVecf<4,2> const* in_xy0,
	Quatf const* in_T_rot,
	Vecf<4> const* in_points,
	Vecf<4> const* in_g_centers,
	Vecf<4> const* in_g_invscale,
	Quatf const* in_g_invrot,
	Matf<12,3>* out_grid_M
)
{
	auto block_idx = blockIdx.x + gridDim.x * blockIdx.y;
	auto transform_idx = blockIdx.y;
	auto point_idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto lane_idx = threadIdx.x & (CUDA_WARP_SIZE - 1);
	auto warp_idx = threadIdx.x / CUDA_WARP_SIZE;

	Vecf<3> x0 = in_xy0[transform_idx].col(0).segment<3>(0);
	Vecf<3> y0 = in_xy0[transform_idx].col(1).segment<3>(0);
	Quatf    q = in_T_rot[transform_idx];

	// Fetch the index of the Gaussian associated to this thread
	int32_t g_idx = -1;
	if (point_idx < N_points) {
		g_idx = in_matchups[point_idx + N_points*transform_idx];
	}

	Matf<12,3> M = Matf<12,3>::Zero();
	if (g_idx >= 0) {
		Matf<3> SR = in_g_invscale[g_idx].segment<3>(0).asDiagonal() * in_g_invrot[g_idx].toRotationMatrix();
		Matf<3> P  = SR.transpose()*SR;
		Vecf<3> u  = in_points[point_idx].segment<3>(0) - x0;
		Vecf<3> Ru = q*u;
		Vecf<3> v  = in_g_centers[g_idx].segment<3>(0) - y0;
		auto    U  = Ru*u.transpose();

		M.block(0,0,3,3) = -P*(Ru - v)*u.transpose();
		M.block(3,0,3,3) = P*skewsym(-U.col(0));
		M.block(6,0,3,3) = P*skewsym(-U.col(1));
		M.block(9,0,3,3) = P*skewsym(-U.col(2));
	}

	// Sum-reduce this warp's matrix
	for (unsigned w = CUDA_WARP_SIZE/2; w; w >>= 1) {
		for (unsigned i = 0; i < 12; i ++) {
			for (unsigned j = 0; j < 3; j ++) {
				M(i,j) += __shfl_xor_sync(UINT32_MAX, M(i,j), w);
			}
		}
	}

	// Store each warp's matrix into sharedmem
	__shared__ Wrapper<decltype(M)> block_M[UPO_CUDA_WARPS_PER_BLOCK];
	if (lane_idx == 0) {
		*block_M[warp_idx] = M;
	}

	// Sum-reduce this thread block's matrix
	for (unsigned w = UPO_CUDA_WARPS_PER_BLOCK/2; w; w >>= 1) {
		__syncthreads();
		if (lane_idx == 0 && warp_idx < w) {
			*block_M[warp_idx] += *block_M[warp_idx+w];
		}
	}

	// Store this thread block's reduced matrix
	if (lane_idx == 0 && warp_idx == 0) {
		out_grid_M[block_idx] = *block_M[0];
	}

}

}

void IcgContext::cuda_matchupP2G(float max_mahal)
{
	icgMatchupP2G_Impl LAUNCH_ARGS(
		m_numPoints,
		m_numGaussians,
		max_mahal*max_mahal,
		m_T_tran,
		m_T_rot,
		m_points,
		m_g_centers,
		m_g_invscale,
		m_g_invrot,
		m_matchups,
		sr_matchOut()
	);
}

void IcgContext::cuda_sumReducePxy()
{
	icgSumReducePxy_Impl LAUNCH_ARGS(
		m_numPoints,
		m_matchups,
		m_points,
		m_g_centers,
		m_g_invscale,
		m_g_invrot,
		sr_pmat(),
		sr_xy0()
	);
}

void IcgContext::cuda_sumReduceRotOpt()
{
	icgSumReduceRotOpt_Impl LAUNCH_ARGS(
		m_numPoints,
		m_matchups,
		m_xy0,
		m_T_rot,
		m_points,
		m_g_centers,
		m_g_invscale,
		m_g_invrot,
		sr_rotopt()
	);
}

}
