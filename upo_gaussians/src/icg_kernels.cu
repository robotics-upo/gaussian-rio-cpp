#include "hip/hip_runtime.h"
#include "gaussian_scan_matching.hpp"

#define LAUNCH_ARGS <<<dim3(m_numBlocks, m_numParticles), UPO_CUDA_NUM_THREADS>>>

namespace upo_gaussians::detail {

namespace {

__global__ void icgMatchupP2G_Impl(
	uint32_t N_points,
	uint32_t N_gaussians,
	float max_sqmahal,
	Vecf<4> const* in_T_tran,
	Quatf const* in_T_rot,
	Vecf<4> const* in_points,
	Vecf<4> const* in_g_centers,
	Vecf<4> const* in_g_invscale,
	Quatf const* in_g_invrot,
	int32_t* out_matchups,
	MatchOutf* out_grid_matchout
)
{
	auto block_idx = blockIdx.x + gridDim.x * blockIdx.y;
	auto transform_idx = blockIdx.y;
	auto point_idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto lane_idx = threadIdx.x & (CUDA_WARP_SIZE - 1);
	auto warp_idx = threadIdx.x / CUDA_WARP_SIZE;

	bool is_active = point_idx < N_points;
	int32_t best_idx = -1;
	float best_sqmahal = is_active ? max_sqmahal : 0.0f;

	Vecf<3> point;
	if (is_active) {
		point = in_T_tran[transform_idx].segment<3>(0) + in_T_rot[transform_idx]*in_points[point_idx].segment<3>(0);
	}

	for (uint32_t base = 0; base < N_gaussians; base += CUDA_WARP_SIZE) {
		unsigned maxfetch = capToWarpSize(N_gaussians - base);

		Vecf<3> warp_g_center;
		Vecf<3> warp_g_invscale;
		Quatf   warp_g_invrot;
		if (lane_idx < maxfetch) {
			warp_g_center   = in_g_centers[base+lane_idx].segment<3>(0);
			warp_g_invscale = in_g_invscale[base+lane_idx].segment<3>(0);
			warp_g_invrot   = in_g_invrot[base+lane_idx];
		}

		for (unsigned i = 0; i < maxfetch; i ++) {
			Vecf<3> g_center;
			Vecf<3> g_invscale;
			Quatf   g_invrot;
			for (unsigned row = 0; row < 3; row ++) {
				g_center(row) = __shfl_sync(UINT32_MAX, warp_g_center(row), i);
				g_invscale(row) = __shfl_sync(UINT32_MAX, warp_g_invscale(row), i);
				g_invrot.coeffs()(row) = __shfl_sync(UINT32_MAX, warp_g_invrot.coeffs()(row), i);
			}
			g_invrot.coeffs()(3) = __shfl_sync(UINT32_MAX, warp_g_invrot.coeffs()(3), i);

			if (!is_active) {
				continue;
			}

			int32_t curidx = base+i;
			auto vector = g_invscale.cwiseProduct(g_invrot*(point - g_center));
			float sqmahal = vector.squaredNorm();

			if (sqmahal < best_sqmahal) {
				best_idx = curidx;
				best_sqmahal = sqmahal;
			}
		}
	}

	if (is_active) {
		out_matchups[point_idx + N_points*transform_idx] = best_idx;
	}

	// Sum-reduce for this warp
	uint32_t matched_mask = __ballot_sync(UINT32_MAX, best_idx >= 0);
	for (unsigned w = CUDA_WARP_SIZE/2; w; w >>= 1) {
		best_sqmahal += __shfl_xor_sync(UINT32_MAX, best_sqmahal, w);
	}

	// Prepare the thread block sum-reduce
	__shared__ float block_sqmahal[UPO_CUDA_WARPS_PER_BLOCK];
	__shared__ int32_t block_matches[UPO_CUDA_WARPS_PER_BLOCK];
	if (lane_idx == 0) {
		block_sqmahal[warp_idx] = best_sqmahal;
		block_matches[warp_idx] = __builtin_popcount(matched_mask);
	}

	// Sum-reduce for this thread block
	for (unsigned w = UPO_CUDA_WARPS_PER_BLOCK/2; w; w >>= 1) {
		__syncthreads();
		if (lane_idx == 0 && warp_idx < w) {
			block_sqmahal[warp_idx] += block_sqmahal[warp_idx+w];
			block_matches[warp_idx] += block_matches[warp_idx+w];
		}
	}

	// Store the reduced value
	if (lane_idx == 0 && warp_idx == 0) {
		auto& out = out_grid_matchout[block_idx];
		out.sqmahal = block_sqmahal[0];
		out.matches = block_matches[0];
	}
}

}

void IcgContext::cuda_matchupP2G(float max_mahal)
{
	icgMatchupP2G_Impl LAUNCH_ARGS(
		m_numPoints,
		m_numGaussians,
		max_mahal*max_mahal,
		m_T_tran,
		m_T_rot,
		m_points,
		m_g_centers,
		m_g_invscale,
		m_g_invrot,
		m_matchups,
		sr_matchOut()
	);
}

}

